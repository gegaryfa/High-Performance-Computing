#include "hip/hip_runtime.h"
/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>
#include <time.h>
#include <math.h>

unsigned int filter_radius;

#define ANSI_COLOR_WHITE   "\x1B[0m"	//WHITE
#define ANSI_COLOR_GREEN   "\x1b[32m"   //GREEN color for GPU messages :)
#define ANSI_COLOR_RED     "\x1b[31m"	//RED


// Define this to turn on error checking
#define CUDA_ERROR_CHECK
 
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.005
#define is_power_of_2(num)	(0 == ((num != 1) && (num & (num - 1))))


/*
 *Switch between double and single precision
 *Default: float
 *Use -USE_DOUBLE=1 as a flag at compilation to use double precision
 */


//#define USE_DOUBLE

#ifdef USE_DOUBLE
typedef double real;
#else
typedef float real;
#endif


//#define TIMING


void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}
 
void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
 
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}





////////////////////////////////////////////////////////////////////////////////
// CPU Functions
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(real *h_Dst, real *h_Src, real *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {	//col
    for (x = 0; x < imageW; x++) {	//row
      real sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

     
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(real *h_Dst, real *h_Src, real *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      real sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// GPU Kernels
////////////////////////////////////////////////////////////////////////////////

/*__global__ void convolutionRowGPU(real *d_Dst,	(out)-->rowConvoled image
 *							real *d_Src, 			(in) -->Image to be convoled
 *							real *d_Filter, 		(in) -->Filter for convolution
 *							int filterR) 			(in) -->Filter Radius
 *
 *
*/
__global__ void convolutionRowGPU(real *d_Dst,
 							real *d_Src, 
							real *d_Filter, 
							int filterR) {

	int ix = blockIdx.x * blockDim.x + threadIdx.x;	//col
	int iy = blockIdx.y * blockDim.y + threadIdx.y;	//row
	
	int grid_width = gridDim.x * blockDim.x; 
	int idx = iy * grid_width + ix;					//destination

	real sum = 0;
	int k;

	for (k = -filterR; k <= filterR; k++) {
		int d = ix + k;

		if (d >= 0 && d < grid_width) {
			sum += d_Src[iy * grid_width + d] * d_Filter[filterR - k];
		}     

		d_Dst[idx] = sum;
	}
      
}


/*__global__ void convolutionColumnGPU(real *d_Dst,	(out)-->rowConvoled image
 *							real *d_Src, 			(in) -->Image to be convoled
 *							real *d_Filter, 		(in) -->Filter for convolution
 *							int filterR) 			(in) -->Filter Radius
 *
 *
*/
__global__ void convolutionColumnGPU(real *d_Dst,
 							real *d_Src, 
							real *d_Filter, 
							int filterR) {

	int ix = blockIdx.x * blockDim.x + threadIdx.x;	//col
	int iy = blockIdx.y * blockDim.y + threadIdx.y;	//row
	
	int grid_width = gridDim.x * blockDim.x; 
	int idx = iy * grid_width + ix;					//destination

	real sum = 0;
	int k;

	for (k = -filterR; k <= filterR; k++) {
		int d = iy + k;

		if (d >= 0 && d < grid_width) {
			sum += d_Src[d * grid_width + ix] * d_Filter[filterR - k];
		}     

		d_Dst[idx] = sum;
	}	

}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
	/*Host pointers (CPU)*/
    real
    *h_Filter = 0,
    *h_Input = 0,
    *h_Buffer = 0,
    *h_OutputCPU = 0,
	*h_OutputGPU = 0;

	/*Device pointers (GPU)*/
    real
    *d_Filter = 0,
    *d_Input = 0,
    *d_Buffer = 0,
    *d_OutputGPU = 0;


    int imageW;
    int imageH;
	int TILE_W = 32;			//TILE_W x TILE_W blocks inside the grid	
    unsigned int i, j;

	struct timespec  tv1, tv2;
	hipEvent_t start, stop;
	float GPU_time = 0.0,
		   GPU_time_transfer = 0.0,
		   GPU_time_calc = 0.0;


	do{
		printf("Enter filter radius : ");
		scanf("%d", &filter_radius);
	}while(filter_radius <=0);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

	do{
		printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
		scanf("%d", &imageW);
	}while(imageW <= FILTER_LENGTH || !is_power_of_2(imageW));

    imageH = imageW;

	unsigned int sizeof_image = imageW * imageH * sizeof(real);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");

	/* Allocate memory for host */
    h_Filter    = (real *)malloc(FILTER_LENGTH * sizeof(real));
    h_Input     = (real *)malloc(sizeof_image);
    h_Buffer    = (real *)malloc(sizeof_image);
    h_OutputCPU = (real *)malloc(sizeof_image);
	h_OutputGPU = (real *)malloc(sizeof_image);

	// if memory allocation on host failed, report an error message
	if(h_Filter == 0 || h_Input == 0 || h_Buffer ==0 || h_OutputCPU == 0 || h_OutputGPU == 0 ){
		printf("couldn't allocate memory\n");
		return 1;
	}
	printf("Allocating host arrays finished...\n\n");

	/* Allocate memory for device */
	printf("Allocating and initializing device arrays...\n");
	hipSetDevice(0);
	CudaSafeCall( hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(real)) );
	CudaSafeCall( hipMalloc((void **)&d_Input,  sizeof_image) );
	CudaSafeCall( hipMalloc((void **)&d_Buffer, sizeof_image) );
	CudaSafeCall( hipMalloc((void **)&d_OutputGPU, sizeof_image) );

	/* Initialize device memory */
	CudaSafeCall( hipMemset(d_Filter,    0.f, FILTER_LENGTH * sizeof(real)) );
	CudaSafeCall( hipMemset(d_Input,     0.f, sizeof_image) );
	CudaSafeCall( hipMemset(d_Buffer,    0.f, sizeof_image) );
	CudaSafeCall( hipMemset(d_OutputGPU, 0.f, sizeof_image) );
	
	printf("Allocating device arrays finished...\n\n");

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (real)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (real)rand() / ((real)RAND_MAX / 255) + (real)rand() / (real)RAND_MAX;
    }


    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

#ifdef TIMING
double cpuTries[12], gpuTries[12],
			cpuTime[10], gpuTime[10];
double average, std_deviation, sum = 0, sum1 = 0,swap, variance;
int c, d;

for(j = 0; j<12; j++){

	/*Get the starting time.*/	
	clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

	/*Take the end time	*/
	clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);

	cpuTries[j]=((double)(
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec)  ) * 1000.0);

}

/*sort*/
	for (c = 0 ; c < ( 11 ); c++){
		for (d = 0 ; d < 12 - c - 1; d++){
			if (cpuTries[d] > cpuTries[d+1]) /* For decreasing order use < */
			{
		    swap = cpuTries[d];
		    cpuTries[d] = cpuTries[d+1];
		    cpuTries[d+1] = swap;
		  }
		}
  	}	
	/*through out max and min*/ /*LATHOS*/
	for(i=1; i<11;i++){
		cpuTime[i-1] = cpuTries[i];
	}

	/*  Compute the sum of all elements */
	for (i = 0; i < 10; i++){
		sum = sum + cpuTime[i];
	}

	average = sum / (double)10;
	/*  Compute  variance  and standard deviation  */
	for (i = 0; i < 10; i++){
		sum1 = sum1 + pow((cpuTime[i] - average), 2);
	}
	variance = sum1 / (float)10;
	std_deviation = sqrt(variance);
	printf("Average in CPU = %.7lf\n", average);
	//printf("variance of all elements = %.5lf\n", variance);
	printf("Standard deviation in CPU = %.7lf\n", std_deviation);

#else
	/*Get the starting time.*/	
	clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

	/*Take the end time	*/
	clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);
#endif


	printf("CPU computation finished...\n\n");



	/*create gpu timers*/
	hipEventCreate(&start);
	hipEventCreate(&stop);


	printf(ANSI_COLOR_GREEN	"GPU computation..." ANSI_COLOR_WHITE "\n");

#ifdef TIMING
for(j = 0; j<12; j++){
sum = 0;
sum1 = 0;
#endif

	/* Copy filter and image to device memory  */
		
	hipEventRecord(start);
	CudaSafeCall( hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(real), hipMemcpyHostToDevice) );
	CudaSafeCall( hipMemcpy(d_Input, h_Input, sizeof_image, hipMemcpyHostToDevice) );
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&GPU_time_transfer, start, stop);

	/* imageH x imageW threads per block */
	dim3 block_size;
	block_size.x = TILE_W;
	block_size.y = TILE_W;

	/* n x n grid*/
	dim3 grid_size;
	grid_size.x = imageW/block_size.x;
	grid_size.y = imageH/block_size.y;

	hipEventRecord(start);
	convolutionRowGPU<<<grid_size,block_size>>>(d_Buffer, d_Input, d_Filter, filter_radius); // convolution kata grammes
	CudaCheckError();
    convolutionColumnGPU<<<grid_size,block_size>>>(d_OutputGPU, d_Buffer, d_Filter, filter_radius); // convolution kata sthles
	CudaCheckError();
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&GPU_time_calc, start, stop);

	/* Copy the result back to host*/
	hipEventRecord(start);
	CudaSafeCall( hipMemcpy(h_OutputGPU, d_OutputGPU, sizeof_image, hipMemcpyDeviceToHost) );
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	GPU_time_transfer += hipEventElapsedTime(&GPU_time_transfer, start, stop);

	GPU_time = GPU_time_transfer + GPU_time_calc;
	
#ifdef TIMING
	gpuTries[j] = GPU_time;
}

/*sort*/
	for (c = 0 ; c < ( 11 ); c++){
		for (d = 0 ; d < 12 - c - 1; d++){
			if (gpuTries[d] > gpuTries[d+1]) /* For decreasing order use < */
			{
		    swap = gpuTries[d];
		    gpuTries[d] = gpuTries[d+1];
		    gpuTries[d+1] = swap;
		  }
		}
  	}	
	/*through out max and min*/ /*LATHOS*/
	for(i=1; i<11;i++){
		gpuTime[i-1] = gpuTries[i];
	}

	/*  Compute the sum of all elements */
	for (i = 0; i < 10; i++){
		sum = sum + gpuTime[i];
	}

	average = sum / (double)10;
	/*  Compute  variance  and standard deviation  */
	for (i = 0; i < 10; i++){
		sum1 = sum1 + pow((gpuTime[i] - average), 2);
	}
	variance = sum1 / (float)10;
	std_deviation = sqrt(variance);
	printf("Average in GPU = %.7lf\n", average);
	//printf("variance of all elements = %.5lf\n", variance);
	printf("Standard deviation in GPU = %.7lf\n", std_deviation);
#endif


	printf(ANSI_COLOR_GREEN	"GPU computation finished..." ANSI_COLOR_WHITE "\n\n");

	bool error_check = false;
	real errorValue = 0;
	//real max_error = 0;
	for(j = 0; j<imageW * imageH; j++){
		if(ABS(h_OutputGPU[j] - h_OutputCPU[j]) > accuracy){
			//max_error = ABS(h_OutputGPU[j] - h_OutputCPU[j]);
			errorValue = ABS(h_OutputGPU[j] - h_OutputCPU[j]);
			error_check = true;			
			break;
		}
	}
	

	printf("\n=============RESULTS=============\n");

	if(error_check == true){
		printf(ANSI_COLOR_RED	"The images differ \n" ANSI_COLOR_WHITE);
		printf(ANSI_COLOR_RED   "for :%f \n" ANSI_COLOR_WHITE, errorValue );	
	}
	else
		printf(ANSI_COLOR_GREEN	"The images are identical \n" ANSI_COLOR_WHITE );


	

	printf("\n==============TIMES==============");

	printf ("\nCPU time %.7g ms\n",(double)(
			(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
			(double) (tv2.tv_sec - tv1.tv_sec)  ) * 1000.0);					//* 1000.0 gia ms

	printf ("\nGPU time %.7g ms\n", GPU_time);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	// make the host wait until the kernel is finished executing before
	// checking for the last CUDA error. otherwise, we won't detect
	// an error if one occurred
#ifdef CUDA_ERROR_CHECK
	hipDeviceSynchronize();									//remove when code has no bugs
#endif
	// this kind of "blocking" operation is usually only appropriate during
	// the debugging process because it forces the host to wait on the device
	// while it could be busy doing other things. once the code has been
	// debugged, frequent error checking code should be eliminated or disabled

	// ask CUDA for the last error to occur (if one exists)

	hipError_t error = hipGetLastError();						
	if(error != hipSuccess){
		// something's gone wrong
		// print out the CUDA error as a string
		printf("CUDA Error: %s\n", hipGetErrorString(error));

		// we can't recover from the error -- exit the program
		return 1;
	}
	// no error occurred, proceed as usual

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  



    // free all the allocated memory
    free(h_OutputGPU);
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);

	CudaSafeCall(hipFree(d_OutputGPU));
	CudaSafeCall(hipFree(d_Buffer));
	CudaSafeCall(hipFree(d_Input));
	CudaSafeCall(hipFree(d_Filter));

    // Do a device reset just in case... 
     hipDeviceReset();


    return 0;
}
