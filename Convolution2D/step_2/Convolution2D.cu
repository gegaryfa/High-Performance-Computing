/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdbool.h>

unsigned int filter_radius;

#define ANSI_COLOR_WHITE   "\x1B[0m"	//WHITE
#define ANSI_COLOR_GREEN   "\x1b[32m"   //GREEN color for GPU messages :)
#define ANSI_COLOR_RED     "\x1b[31m"	//RED


// Define this to turn on error checking
#define CUDA_ERROR_CHECK
 
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.005														//max error =0.005  
#define is_power_of_2(num)	(0 == ((num != 1) && (num & (num - 1))))


/*
 *Switch between double and single precision
 *Default: float
 *Use -USE_DOUBLE=1 as a flag at compilation to use double precision
 */


#ifdef USE_DOUBLE
typedef double real;
#else
typedef float real;
#endif



void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}
 
void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
 
    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif
 
    return;
}





////////////////////////////////////////////////////////////////////////////////
// CPU Functions
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(real *h_Dst, real *h_Src, real *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int x, y, k;
                      
  for (y = 0; y < imageH; y++) {	//col
    for (x = 0; x < imageW; x++) {	//row
      real sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }     

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

     
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(real *h_Dst, real *h_Src, real *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;
  
  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      real sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }   
 
        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// GPU Kernels
////////////////////////////////////////////////////////////////////////////////

/*__global__ void convolutionRowGPU(real *d_Dst,	(out)-->rowConvoled image
 *							real *d_Src, 			(in) -->Image to be convoled
 *							real *d_Filter, 		(in) -->Filter for convolution
 *							int filterR) 			(in) -->Filter Radius
 *
 *
*/
__global__ void convolutionRowGPU(real *d_Dst,
 							real *d_Src, 
							real *d_Filter, 
							int filterR) {

	int ix = blockIdx.x * blockDim.x + threadIdx.x;	//col
	int iy = blockIdx.y * blockDim.y + threadIdx.y;	//row
	int idx = iy * blockDim.x + ix;					//destination

	real sum = 0;
	int k;

	for (k = -filterR; k <= filterR; k++) {
		int d = ix + k;

		if (d >= 0 && d < blockDim.x) {
			sum += d_Src[iy * blockDim.x + d] * d_Filter[filterR - k];
		}     

		d_Dst[idx] = sum;
	}
      
}


/*__global__ void convolutionColumnGPU(real *d_Dst,	(out)-->rowConvoled image
 *							real *d_Src, 			(in) -->Image to be convoled
 *							real *d_Filter, 		(in) -->Filter for convolution
 *							int filterR) 			(in) -->Filter Radius
 *
 *
*/
__global__ void convolutionColumnGPU(real *d_Dst,
 							real *d_Src, 
							real *d_Filter, 
							int filterR) {

	int ix = threadIdx.x + blockDim.x * blockIdx.x;	//col
	int iy = threadIdx.y + blockDim.y * blockIdx.y;	//row
	int idx = iy * blockDim.x + ix;					//destination

	real sum = 0;
	int k;

	for (k = -filterR; k <= filterR; k++) {
		int d = iy + k;

		if (d >= 0 && d < blockDim.x) {
			sum += d_Src[d * blockDim.x + ix] * d_Filter[filterR - k];
		}     

		d_Dst[idx] = sum;
	}	

}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
	/*Host pointers (CPU)*/
    real
    *h_Filter = 0,
    *h_Input = 0,
    *h_Buffer = 0,
    *h_OutputCPU = 0,
	*h_OutputGPU = 0;

	/*Device pointers (GPU)*/
    real
    *d_Filter = 0,
    *d_Input = 0,
    *d_Buffer = 0,
    *d_OutputGPU = 0;


    int imageW;
    int imageH;
    unsigned int i, j;


	do{
		printf("Enter filter radius : ");
		scanf("%d", &filter_radius);
	}while(filter_radius <=0);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

	do{
		printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
		scanf("%d", &imageW);
	}while(imageW <= FILTER_LENGTH || !is_power_of_2(imageW));

    imageH = imageW;

	unsigned int sizeof_image = imageW * imageH * sizeof(real);

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");

	/* Allocate memory for host */
    h_Filter    = (real *)malloc(FILTER_LENGTH * sizeof(real));
    h_Input     = (real *)malloc(sizeof_image);
    h_Buffer    = (real *)malloc(sizeof_image);
    h_OutputCPU = (real *)malloc(sizeof_image);
	h_OutputGPU = (real *)malloc(sizeof_image);

	// if memory allocation on host failed, report an error message
	if(h_Filter == 0 || h_Input == 0 || h_Buffer ==0 || h_OutputCPU == 0 || h_OutputGPU == 0 ){
		printf("couldn't allocate memory\n");
		return 1;
	}

	/* Allocate memory for device */
	printf("Allocating and initializing device arrays...\n");
	hipSetDevice(0);
	CudaSafeCall( hipMalloc((void **)&d_Filter, FILTER_LENGTH * sizeof(real)) );
	CudaSafeCall( hipMalloc((void **)&d_Input,  sizeof_image) );
	CudaSafeCall( hipMalloc((void **)&d_Buffer, sizeof_image) );
	CudaSafeCall( hipMalloc((void **)&d_OutputGPU, sizeof_image) );

	/* Initialize device memory */
	CudaSafeCall( hipMemset(d_Filter,    0.f, FILTER_LENGTH * sizeof(real)) );
	CudaSafeCall( hipMemset(d_Input,     0.f, sizeof_image) );
	CudaSafeCall( hipMemset(d_Buffer,    0.f, sizeof_image) );
	CudaSafeCall( hipMemset(d_OutputGPU, 0.f, sizeof_image) );
	

    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (real)(rand() % 16);
    }

    for (i = 0; i < imageW * imageH; i++) {
        h_Input[i] = (real)rand() / ((real)RAND_MAX / 255) + (real)rand() / (real)RAND_MAX;
    }


    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");

    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

	printf("CPU computation finished...\n");




	printf(ANSI_COLOR_GREEN	"GPU computation..." ANSI_COLOR_WHITE "\n");

	/* Copy filter and image to device memory  */
	CudaSafeCall( hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(real), hipMemcpyHostToDevice) );
	CudaSafeCall( hipMemcpy(d_Input, h_Input, sizeof_image, hipMemcpyHostToDevice) );
	

	/* imageH x imageW threads per block */
	dim3 block_size;
	block_size.x = imageW;
	block_size.y = imageH;

	/* 1 x 1 grid*/
	dim3 grid_size;
	grid_size.x = 1;
	grid_size.y = 1;


	convolutionRowGPU<<<grid_size,block_size>>>(d_Buffer, d_Input, d_Filter, filter_radius); // convolution kata grammes
	CudaCheckError();
    convolutionColumnGPU<<<grid_size,block_size>>>(d_OutputGPU, d_Buffer, d_Filter, filter_radius); // convolution kata sthles
	CudaCheckError();

	/* Copy the result back to host*/
	CudaSafeCall( hipMemcpy(h_OutputGPU, d_OutputGPU, sizeof_image, hipMemcpyDeviceToHost) );
	
	printf(ANSI_COLOR_GREEN	"GPU computation finished..." ANSI_COLOR_WHITE "\n");

	bool error_check = false;
	real errorValue = 0;
	real max_error = 0.0;
	for(j = 0; j<imageW * imageH; j++){
		if(ABS(h_OutputGPU[j] - h_OutputCPU[j]) > accuracy){
			errorValue = ABS(h_OutputGPU[j] - h_OutputCPU[j]);
			error_check = true;			
			//break;
		}
		if(ABS(h_OutputGPU[j] - h_OutputCPU[j]) > max_error){
			max_error = ABS(h_OutputGPU[j] - h_OutputCPU[j]);
		}
	}
	

	printf("\n=============RESULTS=============\n");

	if(error_check == true){
		printf(ANSI_COLOR_RED	"The images differ \n" ANSI_COLOR_WHITE);
		printf(ANSI_COLOR_RED   "for :%.15f \n" ANSI_COLOR_WHITE,errorValue  );
		printf(ANSI_COLOR_RED   "max error =  :%.15f \n" ANSI_COLOR_WHITE,max_error  );	
	}
	else
		printf(ANSI_COLOR_GREEN	"The images are identical \n" ANSI_COLOR_WHITE );


	// make the host wait until the kernel is finished executing before
	// checking for the last CUDA error. otherwise, we won't detect
	// an error if one occurred
	//cudaThreadSynchronize();									//remove when code has no bugs

	// this kind of "blocking" operation is usually only appropriate during
	// the debugging process because it forces the host to wait on the device
	// while it could be busy doing other things. once the code has been
	// debugged, frequent error checking code should be eliminated or disabled

	// ask CUDA for the last error to occur (if one exists)

	hipError_t error = hipGetLastError();						
	if(error != hipSuccess){
		// something's gone wrong
		// print out the CUDA error as a string
		printf("CUDA Error: %s\n", hipGetErrorString(error));

		// we can't recover from the error -- exit the program
		return 1;
	}
	// no error occurred, proceed as usual

    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  



    // free all the allocated memory
    free(h_OutputGPU);
	free(h_OutputCPU);
	free(h_Buffer);
	free(h_Input);
	free(h_Filter);

	hipFree(d_OutputGPU);
	hipFree(d_Buffer);
	hipFree(d_Input);
	hipFree(d_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
     hipDeviceReset();


    return 0;
}
